#include "hip/hip_runtime.h"
#include <float.h>
#include <math.h>
#include <stdio.h>
#include "transforms.cuh"
#include "../include/util.h"

#define max(a, b) ((a) > (b) ? (a) : (b))
#define min(a, b) ((a) < (b) ? (a) : (b))

__global__ void rgb_to_yuv(float *in_buf, float *out_buf, int tot_pixels)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int rgb_idx = tid * 3;
    if (rgb_idx < tot_pixels)
    {
        // in_buf is rgb
        float Y = 0.299 * in_buf[rgb_idx] + 0.587 * in_buf[rgb_idx + 1] + 0.114 * in_buf[rgb_idx + 2];
        float U = 0.492 * (in_buf[rgb_idx + 2] - Y);
        float V = 0.877 * (in_buf[rgb_idx] - Y);
        out_buf[rgb_idx] = Y;
        out_buf[rgb_idx + 1] = U;
        out_buf[rgb_idx + 2] = V;
    }
}

// Min-Reduce algorithm.
__global__ void find_minimum(float *buf, float *buf_r, int stride, int buf_size)
{
    /*
        A simple way would be for each thread to load data from global memory at each of the log(n) step,
        and write back the data. Instead, threads can load data to the shared memory once, and continue with subsequent
        computation without having to read/write to global mem.
    */
    // extern __shared__ double reduced_min[];
    __shared__ float reduced_min[MAX_THREADS_PER_BLOCK];
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;

    if (tidx * stride < buf_size)
    {
        reduced_min[threadIdx.x] = buf[tidx * stride];
    }
    else
    {
        reduced_min[threadIdx.x] = FLT_MAX;
    }
    __syncthreads();

    for (int r = 1; r < blockDim.x; r *= 2)
    {
        int buf_idx = 2 * r * threadIdx.x;
        if (tidx < buf_size && buf_idx < blockDim.x)
        {
            reduced_min[buf_idx] = fmin(reduced_min[buf_idx], reduced_min[buf_idx + r]);
        }
        __syncthreads();
    }
    if (threadIdx.x == 0)
    {
        buf_r[blockIdx.x] = reduced_min[0];
    }
}

// Max-Reduce algorithm.
__global__ void find_maximum(float *buf, float *buf_r, int stride, int buf_size)
{
    /*
        A simple way would be for each thread to load data from global memory at each of the log(n) step,
        and write back the data. Instead, threads can load data to the shared memory once, and continue with subsequent
        computation without having to read/write to global mem.
    */
    // extern __shared__ float reduced_min[];
    __shared__ float reduced_max[MAX_THREADS_PER_BLOCK];
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;

    if (tidx * stride < buf_size)
    {
        reduced_max[threadIdx.x] = buf[tidx * stride];
    }
    else
    {
        reduced_max[threadIdx.x] = FLT_MIN;
    }
    __syncthreads();

    for (int r = 1; r < blockDim.x; r *= 2)
    {
        int buf_idx = 2 * r * threadIdx.x;
        if (buf_idx < blockDim.x - r)
        {
            reduced_max[buf_idx] = fmax(reduced_max[buf_idx], reduced_max[buf_idx + r]);
        }
        __syncthreads();
    }
    if (threadIdx.x == 0)
    {
        buf_r[blockIdx.x] = reduced_max[0];
    }
}

__global__ void compute_image_histogram_naive(const float *img, const int img_size, const int stride, float min_bin_val, float max_bin_val, int *d_bins, const int numBins)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int pix_idx = tid * stride;
    if (pix_idx < img_size)
    {
        int bin = static_cast<int>(img[pix_idx] - min_bin_val) * numBins / (max_bin_val - min_bin_val);
        bin = max(0, min(bin, numBins - 1));
        atomicAdd(&(d_bins[bin]), 1); // Number of global atomic adds scale to size of img buffer
    }
}

__global__ void compute_image_histogram_faster(const float *img, const int img_size, const int stride, float min_bin_val, float max_bin_val, int *d_bins, const int numBins)
{
    extern __shared__ int s_res[];
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (threadIdx.x < numBins)
    {
        s_res[threadIdx.x] = 0;
    }
    __syncthreads();

    int pix_idx = tid * stride;
    if (pix_idx < img_size)
    {
        int bin = static_cast<int>(img[pix_idx] - min_bin_val) * numBins / (max_bin_val - min_bin_val);
        bin = max(0, min(bin, numBins - 1));
        atomicAdd(&(s_res[bin]), 1);
    }

    __syncthreads();

    // Number of global atomic adds equstd::cout << "Image Loaded " << std::endl;als number of bins times the number of threadblocks
    if (threadIdx.x < numBins)
    {
        atomicAdd(&d_bins[threadIdx.x], s_res[threadIdx.x]);
    }
}

__global__ void compute_histogram_distribution(const int *dbins, int *cdf, int numBins)
{
    /*
       A simple way would be for each thread to load data from global memory at each of the log(n) step,
       and write back the data. Instead, threads can load data to the shared memory once, and continue with subsequent
       computation without having to read/write to global mem.
   */
    // extern __shared__ double reduced_min[];
    extern __shared__ int s_dbins[];
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;

    if (tidx < numBins)
    {
        s_dbins[threadIdx.x] = dbins[tidx];
    }
    else
    {
        s_dbins[threadIdx.x] = 0;
    }
    __syncthreads();
    // Should r be limited to numBins?
    for (int r = 1; r < blockDim.x; r *= 2)
    {
        int prev_val = 0;
        if (tidx >= r)
        {
            prev_val = s_dbins[tidx - r];
        }
        __syncthreads();
        s_dbins[threadIdx.x] += prev_val;
        __syncthreads();
    }
    if (tidx < numBins)
    {
        cdf[tidx] = s_dbins[threadIdx.x];
    }
}

extern "C" float *convert_rgb_to_yuv(float *buf, int rows, int cols, int channels)
{
    size_t tot_pixels = rows * cols * channels;
    float *out_buf = (float *)malloc(sizeof(float) * tot_pixels);
    float *cu_inbuf, *cu_outbuf;
    hipMalloc(&cu_inbuf, tot_pixels * sizeof(float));
    hipMalloc(&cu_outbuf, tot_pixels * sizeof(float));
    hipMemcpy(cu_inbuf, buf, tot_pixels * sizeof(float), hipMemcpyHostToDevice);

    int num_blocks = max(1, (tot_pixels + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK);
    rgb_to_yuv<<<num_blocks, MAX_THREADS_PER_BLOCK>>>(cu_inbuf, cu_outbuf, tot_pixels);

    hipMemcpy(out_buf, cu_outbuf, tot_pixels * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(cu_inbuf);
    hipFree(cu_outbuf);
    return out_buf;
}

extern "C" void *get_image_histogram(float *buf, int rows, int cols, int channels, int n_bins)
{
    size_t tot_pixels = rows * cols * channels;
    float *out_buf = (float *)malloc(sizeof(float) * tot_pixels);
    float *cu_inbuf, *cu_outbuf;
    hipMalloc(&cu_inbuf, tot_pixels * sizeof(float));
    hipMalloc(&cu_outbuf, tot_pixels * sizeof(float));
    hipMemcpy(cu_inbuf, buf, tot_pixels * sizeof(float), hipMemcpyHostToDevice);

    int num_blocks = max(1, (tot_pixels + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK);
    rgb_to_yuv<<<num_blocks, MAX_THREADS_PER_BLOCK>>>(cu_inbuf, cu_outbuf, tot_pixels);

    // Compute minimum and maxiumum of luminance channel
    float *min_cu_buf, *max_cu_buf; // buffers to hold min-max values for each thread.
    float min_lum, max_lum;
    hipMalloc(&min_cu_buf, num_blocks * sizeof(float));
    hipMalloc(&max_cu_buf, num_blocks * sizeof(float));
    find_maximum<<<num_blocks, MAX_THREADS_PER_BLOCK>>>(cu_outbuf, max_cu_buf, 3, tot_pixels);
    find_maximum<<<1, MAX_THREADS_PER_BLOCK>>>(max_cu_buf, max_cu_buf, 1, num_blocks);
    find_minimum<<<num_blocks, MAX_THREADS_PER_BLOCK>>>(cu_outbuf, min_cu_buf, 3, tot_pixels);
    find_minimum<<<1, MAX_THREADS_PER_BLOCK>>>(min_cu_buf, min_cu_buf, 1, num_blocks);
    hipMemcpy(&min_lum, min_cu_buf, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&max_lum, max_cu_buf, sizeof(float), hipMemcpyDeviceToHost);
    // Histogram bin
    int *dbins, *cdf, *cdf_cpu;
    cdf_cpu = (int *)malloc(n_bins * sizeof(int));
    hipMalloc(&dbins, n_bins * sizeof(int));
    hipMemset(dbins, 0, n_bins * sizeof(int));
    hipMalloc(&cdf, n_bins * sizeof(int));
    hipMemset(cdf, 0, n_bins * sizeof(int));
    compute_image_histogram_faster<<<num_blocks, MAX_THREADS_PER_BLOCK, n_bins>>>(cu_outbuf, tot_pixels, 3, min_lum, max_lum, dbins, n_bins);
    compute_histogram_distribution<<<1, n_bins, n_bins>>>(dbins, cdf, n_bins);
    hipMemcpy(cdf_cpu, cdf, sizeof(int) * n_bins, hipMemcpyDeviceToHost);
}