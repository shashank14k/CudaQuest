#include "hip/hip_runtime.h"
#include <cassert>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "util.h"

#define N_THREADS 32

__global__ void matrixMul(const double *mat1, const double *mat2, double *mat3, int row1, int col1, int row2, int col2)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < row1 && col < col2)
    {
        double sum = 0;
        for (int i = 0; i < col1; i++)
        {
            sum += mat1[row * col1 + i] * mat2[i * col2 + col];
        }
        mat3[row * col2 + col] = sum;
    }
}

int main(int argc, char *argv[])
{
    if (argc < 4)
    {
        fprintf(stderr, "Error: Required matrix dimensions row1 col1 col2. Usage: ./mat_mul <arg1> <arg2> <arg3>\n");
        exit(EXIT_FAILURE); // Exit the program with a failure status
    }
    int row1 = atoi(argv[1]);
    int col1 = atoi(argv[2]);
    int row2 = atoi(argv[2]);
    int col2 = atoi(argv[3]);

    // Generate random matrices
    double *mat1 = generate_random_double_matrix(row1, col1, 1, 0);
    double *mat2 = generate_random_double_matrix(row2, col2, 1, 0);
    double *mat3 = generate_random_double_matrix(row1, col2, 1, 0);

    double *cu_mat1, *cu_mat2, *cu_mat3;
    hipMalloc(&cu_mat1, row1 * col1 * sizeof(double));
    hipMalloc(&cu_mat2, row2 * col2 * sizeof(double));
    hipMalloc(&cu_mat3, row1 * col2 * sizeof(double));

    hipMemcpy(cu_mat1, mat1, row1 * col1 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(cu_mat2, mat2, row2 * col2 * sizeof(double), hipMemcpyHostToDevice);

    dim3 threads(N_THREADS, N_THREADS);
    dim3 blocks((col2 + threads.x - 1) / threads.x,
                (row1 + threads.y - 1) / threads.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matrixMul<<<blocks, threads>>>(cu_mat1, cu_mat2, cu_mat3, row1, col1, row2, col2);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time_taken = 0;
    hipEventElapsedTime(&time_taken, start, stop);
    printf("Matrix multiplication completed in %f mili-seconds\n", time_taken);
    hipMemcpy(mat3, cu_mat3, row1 * col2 * sizeof(double), hipMemcpyDeviceToHost);
    printf("Verifying results\n");
    verify_cuda_mat_mul(mat1, mat2, mat3, row1, col1, row2, col2);
    printf("MATRIX MULTIPLICATION RAN SUCCESSFULLY AND VERIFIED ON CPU\n");
    // Free memory on device
    hipFree(cu_mat1);
    hipFree(cu_mat2);
    hipFree(cu_mat3);

    free(mat1);
    free(mat2);
    free(mat3);

    return 0;
}